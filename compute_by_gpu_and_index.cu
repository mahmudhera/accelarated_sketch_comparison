#include <iostream>
#include <list>
#include <unordered_map>
#include <vector>
#include <queue>
#include <algorithm>
#include <unordered_set>
#include <fstream>
#include <thread>
#include <mutex>

#include "json.hpp"

#include <zlib.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>


#define CHECK_CUDA(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CHECK_CUBLAS(call) \
    if ((call) != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }


using json = nlohmann::json;

using namespace std;

typedef unsigned long long int hash_t;


std::vector<std::string> sketch_names;
std::vector<std::string> genome_names;
vector<vector<hash_t>> sketches;
int num_sketches;
int num_threads = 1;
unordered_map<hash_t, vector<int>> hash_index;
int count_empty_sketch = 0;
mutex mutex_count_empty_sketch;
float **bit_representation;


void cleanup() {
    for (int i = 0; i < hash_index.size(); i++) {
        delete[] bit_representation[i];
    }
    delete[] bit_representation;
}


void compute_bit_representation() {
    // allocate memory for the bit representation
    size_t num_rows = hash_index.size();
    size_t num_cols = sketches.size();
    bit_representation = new float*[num_rows];
    for (int i = 0; i < num_rows; i++) {
        bit_representation[i] = new float[num_cols];
    }

    // set zeros in the bit representation
    for (int i = 0; i < num_rows; i++) {
        // memset to 0
        memset(bit_representation[i], 0, num_cols * sizeof(float));
    }

    // fill the bit representation
    int row_idx = 0;
    for (auto it = hash_index.begin(); it != hash_index.end(); it++) {
        for (int i = 0; i < it->second.size(); i++) {
            int col_idx = it->second[i];
            bit_representation[row_idx][col_idx] = 1.0;
        }
        row_idx++;
    }
}


void compute_index_from_sketches() {
    
    // create the index using all the hashes
    for (int i = 0; i < sketches.size(); i++) {
        for (int j = 0; j < sketches[i].size(); j++) {
            hash_t hash = sketches[i][j];
            if (hash_index.find(hash) == hash_index.end()) {
                hash_index[hash] = vector<int>();
            }
            hash_index[hash].push_back(i);
        }
    }

    // remove the hashes that only appear in one sketch
    vector<hash_t> hashes_to_remove;
    for (auto it = hash_index.begin(); it != hash_index.end(); it++) {
        if (it->second.size() == 1) {
            hashes_to_remove.push_back(it->first);
        }
    }
    for (int i = 0; i < hashes_to_remove.size(); i++) {
        hash_index.erase(hashes_to_remove[i]);
    }

}


string decompressGzip(const std::string& filename) {
    // Open file
    gzFile file = gzopen(filename.c_str(), "rb");
    if (!file) {
        throw runtime_error("Failed to open gzip file.");
    }

    // Buffer for decompressed data
    const size_t bufferSize = 8192;
    vector<char> buffer(bufferSize);
    string decompressedData;

    int bytesRead;
    while ((bytesRead = gzread(file, buffer.data(), bufferSize)) > 0) {
        decompressedData.append(buffer.data(), bytesRead);
    }

    gzclose(file);
    return decompressedData;
}


pair<std::vector<hash_t>, string> read_min_hashes(const std::string& json_filename) {
    // if filename contains gz
    if (json_filename.find(".gz") != std::string::npos) {
        auto jsonData = json::parse(decompressGzip(json_filename));
        std::vector<hash_t> min_hashes = jsonData[0]["signatures"][0]["mins"];
        std::string genome_name = jsonData[0]["name"];
        return {min_hashes, genome_name};
    }

    // Open the JSON file
    std::ifstream inputFile(json_filename);

    // Check if the file is open
    if (!inputFile.is_open()) {
        std::cerr << "Could not open the file!" << std::endl;
        return {};
    }

    // Parse the JSON data
    json jsonData;
    inputFile >> jsonData;

    // Access and print values
    std::vector<hash_t> min_hashes = jsonData[0]["signatures"][0]["mins"];
    std::string genome_name = jsonData[0]["name"];

    // Close the file
    inputFile.close();

    return {min_hashes, genome_name};
}


void read_sketches_one_chunk(int start_index, int end_index) {
    for (int i = start_index; i < end_index; i++) {
        auto min_hashes_genome_name = read_min_hashes(sketch_names[i]);
        sketches[i] = min_hashes_genome_name.first;
        genome_names[i] = min_hashes_genome_name.second;
        if (sketches[i].size() == 0) {
            mutex_count_empty_sketch.lock();
            count_empty_sketch++;
            mutex_count_empty_sketch.unlock();
        }
    }
}


void read_sketches() {
    for (int i = 0; i < num_sketches; i++) {
        sketches.push_back( vector<hash_t>() );
    }
    // initialize genome_names vector using empty strings
    for (int i = 0; i < num_sketches; i++) {
        genome_names.push_back("");
    }

    int chunk_size = num_sketches / num_threads;
    vector<thread> threads;
    for (int i = 0; i < num_threads; i++) {
        int start_index = i * chunk_size;
        int end_index = (i == num_threads - 1) ? num_sketches : (i + 1) * chunk_size;
        threads.push_back(thread(read_sketches_one_chunk, start_index, end_index));
    }
    for (int i = 0; i < num_threads; i++) {
        threads[i].join();
    }
    // show the number of empty sketches
    cout << "Number of empty sketches: " << count_empty_sketch << endl;

    // show the ids of these empty sketches
    for (int i = 0; i < num_sketches; i++) {
        if (sketches[i].size() == 0) {
            cout << i << " ";
        }
    }
    cout << endl;
}


void get_sketch_names(const std::string& filelist) {
    // the filelist is a file, where each line is a path to a sketch file
    std::ifstream file(filelist);
    if (!file.is_open()) {
        std::cerr << "Could not open the filelist: " << filelist << std::endl;
        return;
    }
    std::string line;
    while (std::getline(file, line)) {
        sketch_names.push_back(line);
    }
    num_sketches = sketch_names.size();
}


int main(int argc, char* argv[]) {
    
    // command line arguments: filelist outputfile
    if (argc != 5) {
        std::cerr << "Usage: " << argv[0] << " <file_list> <out_dir> <num_threads> <gpu_id>" << std::endl;
        return 1;
    }

    auto start_program = std::chrono::high_resolution_clock::now();

    num_threads = std::stoi(argv[3]);

    // get the sketch names
    cout << "Getting sketch names..." << endl;
    get_sketch_names(argv[1]);

    // read the sketches
    cout << "Reading sketches..." << endl;
    read_sketches();
    auto end_read = std::chrono::high_resolution_clock::now();
    std::cout << "Time taken to read the sketches: " << std::chrono::duration_cast<std::chrono::milliseconds>(end_read - start_program).count() << " milliseconds" << std::endl;

    // create the index if needed. otherwise, load from file
    auto start = std::chrono::high_resolution_clock::now();
    cout << "Creating hash index..." << endl;
    compute_index_from_sketches();
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Time taken to create the index: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " milliseconds" << std::endl;

    // create the intersection matrix
    start = std::chrono::high_resolution_clock::now();

    // compute the bit representation
    compute_bit_representation();

    // set the device using the gpu_id
    CHECK_CUDA(hipSetDevice(std::stoi(argv[4])));

    // allocate memory on device
    float *d_bit_representation;
    CHECK_CUDA(hipMalloc(&d_bit_representation, num_sketches * hash_index.size() * sizeof(float)));

    // copy the bit representation to device
    CHECK_CUDA(hipMemcpy(d_bit_representation, bit_representation[0], num_sketches * hash_index.size() * sizeof(float), hipMemcpyHostToDevice));

    // create a cublas handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // create a matrix of size num_rows x num_rows to store the dot products
    float *h_C;
    h_C = new float[num_sketches * num_sketches];
    float *d_C;
    CHECK_CUDA(hipMalloc(&d_C, num_sketches * num_sketches * sizeof(float)));

    // compute bit_representation * bit_representation^T
    float alpha = 1.0;
    float beta = 0.0;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, num_sketches, num_sketches, hash_index.size(), &alpha, d_bit_representation, num_sketches, d_bit_representation, num_sketches, &beta, d_C, num_sketches));

    // copy the result back to host
    CHECK_CUDA(hipMemcpy(h_C, d_C, num_sketches * num_sketches * sizeof(float), hipMemcpyDeviceToHost));

    // free CUDA memory
    CHECK_CUDA(hipFree(d_bit_representation));
    CHECK_CUDA(hipFree(d_C));

    // destroy the handle
    CHECK_CUBLAS(hipblasDestroy(handle));

    // show first 10x10 of the intersection matrix
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            cout << h_C[i * num_sketches + j] << " ";
        }
        cout << endl;
    }

    
    end = std::chrono::high_resolution_clock::now();
    std::cout << "Time taken to create the intersection matrix: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " milliseconds" << std::endl;

    // show time takes for processing only
    std::cout << "Time taken for processing: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - end_read).count() << " milliseconds" << std::endl;
    std::cout << "Time taken overall: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start_program).count() << " milliseconds" << std::endl;

    // cleanup
    cleanup();
    
    return 0;

}