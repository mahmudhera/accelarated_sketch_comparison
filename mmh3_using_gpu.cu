#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include <cstring>
#include <ctime>
#include <unistd.h>
#include <string>
#include <fstream>

using namespace std;

__device__ uint64_t rotateLeft(uint64_t x, int r)
{
    return (x << r) | (x >> (64 - r));
}

__device__ uint64_t getblock64(const uint64_t *p, int i)
{
    uint8_t *pp = (uint8_t *)p;
    // access 8 bytes at a time, then convert to 64-bit integer
    return ((uint64_t)pp[i * 8 + 0] << 0) | ((uint64_t)pp[i * 8 + 1] << 8) | ((uint64_t)pp[i * 8 + 2] << 16) | ((uint64_t)pp[i * 8 + 3] << 24) | ((uint64_t)pp[i * 8 + 4] << 32) | ((uint64_t)pp[i * 8 + 5] << 40) | ((uint64_t)pp[i * 8 + 6] << 48) | ((uint64_t)pp[i * 8 + 7] << 56);
}

__device__ uint64_t fmix64(uint64_t k)
{
    k ^= k >> 33;
    k *= 0xff51afd7ed558ccd;
    k ^= k >> 33;
    k *= 0xc4ceb9fe1a85ec53;
    k ^= k >> 33;
    return k;
}

__device__ void murmurhash3_x64_128(const void *key, const int len, const uint32_t seed, void *out)
{

    const uint8_t *data = (const uint8_t *)key;
    const int nblocks = len / 16;

    uint64_t h1 = seed;
    uint64_t h2 = seed;

    const uint64_t c1 = 0x87c37b91114253d5;
    const uint64_t c2 = 0x4cf5ad432745937f;

    const uint64_t *blocks = (const uint64_t *)(data);

    for (int i = 0; i < nblocks; i++)
    {
        uint64_t k1 = getblock64(blocks, i * 2 + 0);
        uint64_t k2 = getblock64(blocks, i * 2 + 1);

        k1 *= c1;
        k1 = rotateLeft(k1, 31);
        k1 *= c2;
        h1 ^= k1;

        h1 = rotateLeft(h1, 27);
        h1 += h2;
        h1 = h1 * 5 + 0x52dce729;

        k2 *= c2;
        k2 = rotateLeft(k2, 33);
        k2 *= c1;
        h2 ^= k2;

        h2 = rotateLeft(h2, 31);
        h2 += h1;
        h2 = h2 * 5 + 0x38495ab5;
    }

    const uint8_t *tail = (const uint8_t *)(data + nblocks * 16);

    uint64_t k1 = 0;
    uint64_t k2 = 0;

    switch (len & 15)
    {
    case 15:
        k2 ^= ((uint64_t)tail[14]) << 48;
    case 14:
        k2 ^= ((uint64_t)tail[13]) << 40;
    case 13:
        k2 ^= ((uint64_t)tail[12]) << 32;
    case 12:
        k2 ^= ((uint64_t)tail[11]) << 24;
    case 11:
        k2 ^= ((uint64_t)tail[10]) << 16;
    case 10:
        k2 ^= ((uint64_t)tail[9]) << 8;
    case 9:
        k2 ^= ((uint64_t)tail[8]) << 0;
        k2 *= c2;
        k2 = rotateLeft(k2, 33);
        k2 *= c1;
        h2 ^= k2;

    case 8:
        k1 ^= ((uint64_t)tail[7]) << 56;
    case 7:
        k1 ^= ((uint64_t)tail[6]) << 48;
    case 6:
        k1 ^= ((uint64_t)tail[5]) << 40;
    case 5:
        k1 ^= ((uint64_t)tail[4]) << 32;
    case 4:
        k1 ^= ((uint64_t)tail[3]) << 24;
    case 3:
        k1 ^= ((uint64_t)tail[2]) << 16;
    case 2:
        k1 ^= ((uint64_t)tail[1]) << 8;
    case 1:
        k1 ^= ((uint64_t)tail[0]) << 0;
        k1 *= c1;
        k1 = rotateLeft(k1, 31);
        k1 *= c2;
        h1 ^= k1;
    };

    h1 ^= len;
    h2 ^= len;

    h1 += h2;
    h2 += h1;

    h1 = fmix64(h1);
    h2 = fmix64(h2);

    h1 += h2;
    h2 += h1;

    ((uint64_t *)out)[0] = h1;
    ((uint64_t *)out)[1] = h2;
}

// Kernel function
__global__ void hashKernel(const void *input_string, int k, uint32_t seed, void *out, int num_kmers)
{
    // get the index of the thread, linear index of the thread in the thread block
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= num_kmers)
    {
        return;
    }
    murmurhash3_x64_128((char *)input_string + i, k, seed, (uint64_t *)out + 2 * i);
    
}

// Host function to allocate memory and copy data
// arguments: input_string, input_string_length, seed, out, k
void hashOnGPU(const void *input_string, int input_string_length, uint32_t seed, void *out, int k)
{
    int num_kmers = input_string_length - k + 1;
    // num_kmers = 2;

    // allocate memory on device
    void *d_key;
    void *d_out;

    double time_snap = clock();

    hipMalloc(&d_key, input_string_length);
    hipMalloc(&d_out, sizeof(uint64_t) * 2 * num_kmers); // two 64-bit integers for each k-mer

    double time_snap2 = clock();
    std::cout << "Time taken for memory allocation: " << (time_snap2 - time_snap) / CLOCKS_PER_SEC << std::endl;

    // copy data to device
    hipMemcpy(d_key, input_string, input_string_length, hipMemcpyHostToDevice);

    double time_snap3 = clock();
    std::cout << "Time taken for copying data to device: " << (time_snap3 - time_snap2) / CLOCKS_PER_SEC << std::endl;

    // determine the number of threads per block, number of blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_kmers + threadsPerBlock - 1) / threadsPerBlock;

    // measure start and end time
    double start_time = clock();

    // call kernel function
    hashKernel <<<blocksPerGrid, threadsPerBlock>>> (d_key, k, seed, d_out, num_kmers);

    // wait for the kernel to finish
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    }

    double end_time = clock();

    std::cout << "Time taken for the kernel to run: " << (end_time - start_time) / CLOCKS_PER_SEC << std::endl;

    // copy data back to host
    hipMemcpy(out, d_out, sizeof(uint64_t) * 2 * num_kmers, hipMemcpyDeviceToHost);

    double time_snap4 = clock();
    std::cout << "Time taken for copying data back to host: " << (time_snap4 - end_time) / CLOCKS_PER_SEC << std::endl;

    // free memory
    hipFree(d_key);
    hipFree(d_out);

    double time_snap5 = clock();
    std::cout << "Time taken for freeing memory: " << (time_snap5 - time_snap4) / CLOCKS_PER_SEC << std::endl;
}

void readFASTA(const std::string &filename, std::string &header, std::string &sequence)
{
    std::ifstream infile(filename);
    if (!infile)
    {
        std::cerr << "Error opening file: " << filename << std::endl;
        return;
    }

    std::string line;
    while (std::getline(infile, line))
    {
        if (line[0] == '>')
        {
            header = line; // Store the header line
        }
        else
        {
            sequence += line; // Append to the sequence string
        }
    }

    infile.close();
}

int main(int argc, char *argv[])
{
    // first command line argument is the fasta filename
    if (argc != 3)
    {
        std::cerr << "Usage: " << argv[0] << " <in_filename> <out_filename>" << std::endl;
        return 1;
    }

    std::string filename = argv[1];
    std::string out_filename = argv[2];
    std::string header;
    std::string sequence;

    readFASTA(filename, header, sequence);

    const char *input_string = sequence.c_str();
    int input_string_length = strlen(input_string);
    uint32_t seed = 0;
    int k = 21;
    int num_kmers = input_string_length - k + 1;

    uint64_t *out = new uint64_t[2 * num_kmers];

    hashOnGPU(input_string, input_string_length, seed, out, k);
    // wait for the kernel to finish
    hipDeviceSynchronize();

    /*
    std::ofstream outfile(out_filename);
    for (int i = 0; i < num_kmers; i++)
    {
        string kmer = sequence.substr(i, k);
        outfile << kmer << " " << out[2 * i] << " " << out[2 * i + 1] << std::endl;
    }
    */

    delete[] out;

    return 0;
}
