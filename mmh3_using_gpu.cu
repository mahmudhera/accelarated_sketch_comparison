#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include <cstring>
#include <ctime>
#include <unistd.h>
#include <string>

using namespace std;


__device__ uint64_t rotateLeft(uint64_t x, int r) {
    return (x << r) | (x >> (64 - r));
}

__device__ uint64_t getblock64(const uint64_t* p, int i) {
    return p[i];
}

__device__ uint64_t fmix64(uint64_t k) {
    k ^= k >> 33;
    k *= 0xff51afd7ed558ccd;
    k ^= k >> 33;
    k *= 0xc4ceb9fe1a85ec53;
    k ^= k >> 33;
    return k;
}

__device__ void murmurhash3_x64_128(const void* key, const int len, const uint32_t seed, void* out) {
    const uint8_t* data = (const uint8_t*)key;
    const int nblocks = len / 16;

    uint64_t h1 = seed;
    uint64_t h2 = seed;

    const uint64_t c1 = 0x87c37b91114253d5;
    const uint64_t c2 = 0x4cf5ad432745937f;

    const uint64_t* blocks = (const uint64_t*)(data);

    for (int i = 0; i < nblocks; i++) {
        uint64_t k1 = getblock64(blocks, i * 2 + 0);
        uint64_t k2 = getblock64(blocks, i * 2 + 1);

        k1 *= c1;
        k1 = rotateLeft(k1, 31);
        k1 *= c2;
        h1 ^= k1;

        h1 = rotateLeft(h1, 27);
        h1 += h2;
        h1 = h1 * 5 + 0x52dce729;

        k2 *= c2;
        k2 = rotateLeft(k2, 33);
        k2 *= c1;
        h2 ^= k2;

        h2 = rotateLeft(h2, 31);
        h2 += h1;
        h2 = h2 * 5 + 0x38495ab5;
    }

    const uint8_t* tail = (const uint8_t*)(data + nblocks * 16);

    uint64_t k1 = 0;
    uint64_t k2 = 0;

    switch (len & 15) {
    case 15: k2 ^= ((uint64_t)tail[14]) << 48;
    case 14: k2 ^= ((uint64_t)tail[13]) << 40;
    case 13: k2 ^= ((uint64_t)tail[12]) << 32;
    case 12: k2 ^= ((uint64_t)tail[11]) << 24;
    case 11: k2 ^= ((uint64_t)tail[10]) << 16;
    case 10: k2 ^= ((uint64_t)tail[9]) << 8;
    case 9: k2 ^= ((uint64_t)tail[8]) << 0;
        k2 *= c2;
        k2 = rotateLeft(k2, 33);
        k2 *= c1;
        h2 ^= k2;

    case 8: k1 ^= ((uint64_t)tail[7]) << 56;
    case 7: k1 ^= ((uint64_t)tail[6]) << 48;
    case 6: k1 ^= ((uint64_t)tail[5]) << 40;
    case 5: k1 ^= ((uint64_t)tail[4]) << 32;
    case 4: k1 ^= ((uint64_t)tail[3]) << 24;
    case 3: k1 ^= ((uint64_t)tail[2]) << 16;
    case 2: k1 ^= ((uint64_t)tail[1]) << 8;
    case 1: k1 ^= ((uint64_t)tail[0]) << 0;
        k1 *= c1;
        k1 = rotateLeft(k1, 31);
        k1 *= c2;
        h1 ^= k1;
    };

    h1 ^= len;
    h2 ^= len;

    h1 += h2;
    h2 += h1;

    h1 = fmix64(h1);
    h2 = fmix64(h2);

    h1 += h2;
    h2 += h1;

    ((uint64_t*)out)[0] = h1;
    ((uint64_t*)out)[1] = h2;
}

// Kernel function
__global__ void hashKernel(const void* input_string, int k, uint32_t seed, void* out) {
    // get the index of the thread, linear index of the thread in the thread block
    int i = threadIdx.x;
    murmurhash3_x64_128((char*)input_string + i, k, seed, (uint64_t*)out + 2*i);
}

// Host function to allocate memory and copy data
// arguments: input_string, input_string_length, seed, out, k
void hashOnGPU(const void* input_string, int input_string_length, uint32_t seed, void* out, int k) {
    int num_kmers = input_string_length - k + 1;
    num_kmers = 1;

    // allocate memory on device
    void* d_key;
    void* d_out;
    hipMalloc(&d_key, input_string_length);
    hipMalloc(&d_out, sizeof(uint64_t) * 2 * num_kmers); // two 64-bit integers for each k-mer

    // copy data to device
    hipMemcpy(d_key, input_string, input_string_length, hipMemcpyHostToDevice);

    // measure start and end time
    double start_time = clock();

    // call kernel function
    hashKernel<<<1, num_kmers>>>(d_key, k, seed, d_out);

    double end_time = clock();

    std::cout << "Time taken: " << (end_time - start_time) / CLOCKS_PER_SEC << std::endl;

    // copy data back to host
    hipMemcpy(out, d_out, sizeof(uint64_t) * 2 * num_kmers, hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_key);
    hipFree(d_out);
}

int main() {
    const char* input_string = "ACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAGACGTGCAG";
    int input_string_length = strlen(input_string);
    uint32_t seed = 0;
    int k = 21;
    int num_kmers = input_string_length - k + 1;
    num_kmers = 1;
    uint64_t out[2*num_kmers];

    hashOnGPU(input_string, input_string_length, seed, out, k);
    // wait for the kernel to finish
    hipDeviceSynchronize();

    string s(input_string);
    for (int i = 0; i < num_kmers; i++) {
        string kmer = s.substr(i, k);
        std::cout << kmer << " " << out[2*i] << " " << out[2*i + 1] << std::endl;
    }

    return 0;
}
